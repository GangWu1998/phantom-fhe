#include "hip/hip_runtime.h"
#include "cnn_phantom.h"

TensorCipher::TensorCipher()
{
	k_ = 0;
	h_ = 0;
	w_ = 0;
	c_ = 0;
	t_ = 0;
	p_ = 0;
}
TensorCipher::TensorCipher(int logn, int k, int h, int w, int c, int t, int p, vector<double> data, CKKSEvaluator &evaluator, int logp)
{
	if (k != 1)
		throw std::invalid_argument("supported k is only 1 right now");

	// 1 <= logn <= 16
	if (logn < 1 || logn > 16)
		throw std::out_of_range("the value of logn is out of range");
	if (data.size() > static_cast<long unsigned int>(1 << logn))
		throw std::out_of_range("the size of data is larger than n");

	this->k_ = k;
	this->h_ = h;
	this->w_ = w;
	this->c_ = c;
	this->t_ = t;
	this->p_ = p;
	this->logn_ = logn;

	// generate vector that contains data
	vector<double> vec;
	for (int i = 0; i < static_cast<int>(data.size()); i++)
		vec.emplace_back(data[i]);
	for (int i = data.size(); i < 1 << logn; i++)
		vec.emplace_back(0); // zero padding

	// vec size = n
	if (vec.size() != static_cast<long unsigned int>(1 << logn))
		throw std::out_of_range("the size of vec is not n");

	// encode & encrypt
	PhantomPlaintext plain;
	PhantomCiphertext cipher;
	double scale = pow(2.0, logp);
	evaluator.encoder.encode(vec, scale, plain);
	evaluator.encryptor.encrypt(plain, cipher);
	this->set_ciphertext(cipher);
}
TensorCipher::TensorCipher(int logn, int k, int h, int w, int c, int t, int p, PhantomCiphertext cipher)
{
	this->k_ = k;
	this->h_ = h;
	this->w_ = w;
	this->c_ = c;
	this->t_ = t;
	this->p_ = p;
	this->logn_ = logn;
	this->cipher_ = cipher;
}
int TensorCipher::k() const
{
	return k_;
}
int TensorCipher::h() const
{
	return h_;
}
int TensorCipher::w() const
{
	return w_;
}
int TensorCipher::c() const
{
	return c_;
}
int TensorCipher::t() const
{
	return t_;
}
int TensorCipher::p() const
{
	return p_;
}
int TensorCipher::logn() const
{
	return logn_;
}
PhantomCiphertext TensorCipher::cipher() const
{
	return cipher_;
}
void TensorCipher::set_ciphertext(PhantomCiphertext cipher)
{
	cipher_ = cipher;
}
void TensorCipher::print_parms()
{
	cout << "k: " << k_ << endl;
	cout << "h: " << h_ << endl;
	cout << "w: " << w_ << endl;
	cout << "c: " << c_ << endl;
	cout << "t: " << t_ << endl;
	cout << "p: " << p_ << endl;
}
// void multiplexed_parallel_convolution_print(const TensorCipher &cnn_in, TensorCipher &cnn_out, int co, int st, int fh, int fw, const vector<double> &data, vector<double> running_var, vector<double> constant_weight, double epsilon, CKKSEncoder &encoder, Encryptor &encryptor, Evaluator &evaluator, GaloisKeys &gal_keys, vector<PhantomCiphertext> &cipher_pool, ofstream &output, Decryptor &decryptor, SEALContext &context, size_t stage, bool end)
void multiplexed_parallel_convolution_print(const TensorCipher &cnn_in, TensorCipher &cnn_out, int co, int st, int fh, int fw, const vector<double> &data, vector<double> running_var, vector<double> constant_weight, double epsilon, CKKSEvaluator &ckksevaluator, vector<PhantomCiphertext> &cipher_pool, ofstream &output, size_t stage, bool end)
{
	cout << "multiplexed parallel convolution..." << endl;
	output << "multiplexed parallel convolution..." << endl;
	int logn = cnn_in.logn();
	chrono::high_resolution_clock::time_point time_start, time_end;
	chrono::microseconds time_diff;

	time_start = chrono::high_resolution_clock::now();
	// convolution_seal_sparse(cnn_in, cnn_out, hprime, st, kernel, false, data, running_var, constant_weight, epsilon, encoder, encryptor, scale_evaluator, gal_keys, cipher_pool, end);
	multiplexed_parallel_convolution_phantom(cnn_in, cnn_out, co, st, fh, fw, data, running_var, constant_weight, epsilon, ckksevaluator, cipher_pool, end);
	time_end = chrono::high_resolution_clock::now();
	time_diff = chrono::duration_cast<chrono::milliseconds>(time_end - time_start);
	cout << "time : " << time_diff.count() / 1000 << " ms" << endl;
	// cout << "convolution " << stage << " result" << endl;
	output << "time : " << time_diff.count() / 1000 << " ms" << endl;
	// output << "convolution " << stage << " result" << endl;
	auto decry_and_print_temp = cnn_out.cipher();
	decrypt_and_print(decry_and_print_temp, ckksevaluator, 1 << logn, 256, 2);
	cnn_out.print_parms();
	// decrypt_and_print_txt(cnn_out.cipher(), decryptor, encoder, 1<<logn, 256, 2, output); cnn_out.print_parms();
	cout << "remaining level : " << cnn_out.cipher().chain_index() << endl;
	cout << "scale: " << cnn_out.cipher().scale() << endl
		 << endl;
	output << "remaining level : " << cnn_out.cipher().chain_index() << endl;
	output << "scale: " << cnn_out.cipher().scale() << endl
		   << endl;
}
void multiplexed_parallel_batch_norm_phantom_print(const TensorCipher &cnn_in, TensorCipher &cnn_out, vector<double> bias, vector<double> running_mean, vector<double> running_var, vector<double> weight, double epsilon, CKKSEvaluator &ckksevaluator, double B, ofstream &output, size_t stage, bool end)
{
	cout << "multiplexed parallel batch normalization..." << endl;
	output << "multiplexed parallel batch normalization..." << endl;
	int logn = cnn_in.logn();
	chrono::high_resolution_clock::time_point time_start, time_end;
	chrono::microseconds time_diff;

	// batch norm
	time_start = chrono::high_resolution_clock::now();
	multiplexed_parallel_batch_norm_phantom(cnn_in, cnn_out, bias, running_mean, running_var, weight, epsilon, ckksevaluator, B, end);
	time_end = chrono::high_resolution_clock::now();
	time_diff = chrono::duration_cast<chrono::milliseconds>(time_end - time_start);
	cout << "time : " << time_diff.count() / 1000 << " ms" << endl;
	// cout << "batch normalization " << stage << " result" << endl;
	output << "time : " << time_diff.count() / 1000 << " ms" << endl;
	// output << "batch normalization " << stage << " result" << endl;
	auto decry_and_print_temp = cnn_out.cipher();
	decrypt_and_print(decry_and_print_temp, ckksevaluator, 1 << logn, 256, 2);
	cnn_out.print_parms();
	// decrypt_and_print_txt(cnn_out.cipher(), decryptor, encoder, 1<<logn, 256, 2, output); cnn_out.print_parms();
	cout << "remaining level : " << cnn_out.cipher().chain_index() << endl;
	cout << "scale: " << cnn_out.cipher().scale() << endl
		 << endl;
	output << "remaining level : " << cnn_out.cipher().chain_index()<< endl;
	output << "scale: " << cnn_out.cipher().scale() << endl
		   << endl;
}
void approx_ReLU_phantom_print(const TensorCipher &cnn_in, TensorCipher &cnn_out, long comp_no, vector<int> deg, long alpha, vector<Tree> &tree, double scaled_val, long scalingfactor, CKKSEvaluator &ckksevaluator, double B, ofstream &output, size_t stage)
{
	cout << "approximate ReLU..." << endl;
	output << "approximate ReLU..." << endl;
	int logn = cnn_in.logn();
	chrono::high_resolution_clock::time_point time_start, time_end;
	chrono::microseconds time_diff;

	time_start = chrono::high_resolution_clock::now();
	ReLU_phantom(cnn_in, cnn_out, comp_no, deg, alpha, tree, scaled_val, scalingfactor, ckksevaluator, B);
	// ReLU_remove_imaginary_seal(cnn_in, cnn_out, comp_no, deg, alpha, tree, scaled_val, scalingfactor, encryptor, evaluator, scale_evaluator, decryptor, encoder, public_key, secret_key, relin_keys, output, gal_keys, B);
	time_end = chrono::high_resolution_clock::now();
	time_diff = chrono::duration_cast<chrono::milliseconds>(time_end - time_start);
	cout << "time : " << time_diff.count() / 1000 << " ms" << endl;
	// cout << "ReLU function " << stage << " result" << endl;
	output << "time : " << time_diff.count() / 1000 << " ms" << endl;
	// output << "ReLU function " << stage << " result" << endl;
	auto decry_and_print_temp = cnn_out.cipher();
	decrypt_and_print(decry_and_print_temp, ckksevaluator, 1 << logn, 256, 2);
	cnn_out.print_parms();
	// decrypt_and_print_txt(cnn_out.cipher(), decryptor, encoder, 1<<logn, 256, 2, output); cnn_out.print_parms();
	cout << "remaining level : " << cnn_out.cipher().chain_index() << endl;
	cout << "scale: " << cnn_out.cipher().scale() << endl
		 << endl;
	output << "remaining level : " << cnn_out.cipher().chain_index() << endl;
	output << "scale: " << cnn_out.cipher().scale() << endl
		   << endl;

	cout << "intermediate decrypted values: " << endl;
	output << "intermediate decrypted values: " << endl;
	decry_and_print_temp = cnn_out.cipher();
	decrypt_and_print_txt(decry_and_print_temp, ckksevaluator, 1 << logn, 4, 1, output); // cnn_out.print_parms();
}
void bootstrap_print(const TensorCipher &cnn_in, TensorCipher &cnn_out, Bootstrapper &bootstrapper, CKKSEvaluator &ckksevaluator, ofstream &output, size_t stage)
{
	cout << "bootstrapping..." << endl;
	output << "bootstrapping..." << endl;
	PhantomCiphertext ctxt, rtn;
	int logn = cnn_in.logn();
	chrono::high_resolution_clock::time_point time_start, time_end;
	chrono::microseconds time_diff;

	ctxt = cnn_in.cipher();
	time_start = chrono::high_resolution_clock::now();
	// bootstrapper.bootstrap_3(rtn, ctxt);
	bootstrapper.bootstrap_real_3(rtn, ctxt);
	time_end = chrono::high_resolution_clock::now();
	time_diff = chrono::duration_cast<chrono::milliseconds>(time_end - time_start);
	cout << "time : " << time_diff.count() / 1000 << " ms" << endl;
	output << "time : " << time_diff.count() / 1000 << " ms" << endl;
	cnn_out.set_ciphertext(rtn);
	cout << "bootstrapping " << stage << " result" << endl;
	output << "bootstrapping " << stage << " result" << endl;
	auto decry_and_print_temp = cnn_out.cipher();
	decrypt_and_print(decry_and_print_temp, ckksevaluator, 1 << logn, 256, 2);
	cnn_out.print_parms();
	// decrypt_and_print_txt(cnn_out.cipher(), decryptor, encoder, 1<<logn, 256, 2, output); cnn_out.print_parms();
	cout << "remaining level : " << cnn_out.cipher().chain_index() << endl;
	cout << "scale: " << cnn_out.cipher().scale() << endl
		 << endl;
	output << "remaining level : " << cnn_out.cipher().chain_index() << endl;
	output << "scale: " << cnn_out.cipher().scale() << endl
		   << endl;
}


void multiplexed_parallel_downsampling_phantom_print(const TensorCipher &cnn_in, TensorCipher &cnn_out, CKKSEvaluator &ckksevaluator, ofstream &output)
{
	cout << "multiplexed parallel downsampling..." << endl;
	output << "multiplexed parallel downsampling..." << endl;
	int logn = cnn_in.logn();
	chrono::high_resolution_clock::time_point time_start, time_end;
	chrono::microseconds time_diff;

	time_start = chrono::high_resolution_clock::now();
	multiplexed_parallel_downsampling_phantom(cnn_in, cnn_out, ckksevaluator);
	time_end = chrono::high_resolution_clock::now();
	time_diff = chrono::duration_cast<chrono::milliseconds>(time_end - time_start);
	cout << "time : " << time_diff.count() / 1000 << " ms" << endl;
	output << "time : " << time_diff.count() / 1000 << " ms" << endl;
	auto decry_and_print_temp = cnn_out.cipher();
	decrypt_and_print(decry_and_print_temp, ckksevaluator, 1 << logn, 256, 2);
	cnn_out.print_parms();
	// decrypt_and_print_txt(cnn_out.cipher(), decryptor, encoder, 1<<logn, 256, 2, output); cnn_out.print_parms();
	cout << "remaining level : " << cnn_out.cipher().chain_index() << endl;
	cout << "scale: " << cnn_out.cipher().scale() << endl
		 << endl;
	output << "remaining level : " << cnn_out.cipher().chain_index() << endl;
	output << "scale: " << cnn_out.cipher().scale() << endl
		   << endl;
}
void averagepooling_phantom_scale_print(const TensorCipher &cnn_in, TensorCipher &cnn_out, CKKSEvaluator &ckksevaluator, double B, ofstream &output)
{
	cout << "average pooling..." << endl;
	output << "average pooling..." << endl;
	int logn = cnn_in.logn();
	chrono::high_resolution_clock::time_point time_start, time_end;
	chrono::microseconds time_diff;

	time_start = chrono::high_resolution_clock::now();
	// averagepooling_seal_scale(cnn_in, cnn_out, scale_evaluator, gal_keys, B);
	averagepooling_phantom_scale(cnn_in, cnn_out, ckksevaluator, B, output);
	time_end = chrono::high_resolution_clock::now();
	time_diff = chrono::duration_cast<chrono::milliseconds>(time_end - time_start);
	cout << "time : " << time_diff.count() / 1000 << " ms" << endl;
	output << "time : " << time_diff.count() / 1000 << " ms" << endl;
	auto decry_and_print_temp = cnn_out.cipher();
	decrypt_and_print(decry_and_print_temp, ckksevaluator, 1 << logn, 256, 2);
	cnn_out.print_parms();
	// decrypt_and_print_txt(cnn_out.cipher(), decryptor, encoder, 1<<logn, 256, 2, output); cnn_out.print_parms();
	cout << "remaining level : " << cnn_out.cipher().chain_index() << endl;
	cout << "scale: " << cnn_out.cipher().scale() << endl
		 << endl;
	output << "remaining level : " << cnn_out.cipher().chain_index()<< endl;
	output << "scale: " << cnn_out.cipher().scale() << endl
		   << endl;
}
void fully_connected_phantom_print(const TensorCipher &cnn_in, TensorCipher &cnn_out, vector<double> matrix, vector<double> bias, int q, int r, CKKSEvaluator &ckksevaluator, ofstream &output)
{
	cout << "fully connected layer..." << endl;
	output << "fully connected layer..." << endl;
	int logn = cnn_in.logn();
	chrono::high_resolution_clock::time_point time_start, time_end;
	chrono::microseconds time_diff;

	time_start = chrono::high_resolution_clock::now();
	matrix_multiplication_phantom(cnn_in, cnn_out, matrix, bias, q, r,ckksevaluator);
	time_end = chrono::high_resolution_clock::now();
	time_diff = chrono::duration_cast<chrono::milliseconds>(time_end - time_start);
	cout << "time : " << time_diff.count() / 1000 << " ms" << endl;
	output << "time : " << time_diff.count() / 1000 << " ms" << endl;
	auto decry_and_print_temp = cnn_out.cipher();
	decrypt_and_print(decry_and_print_temp, ckksevaluator, 1 << logn, 256, 2);
	cnn_out.print_parms();
	// decrypt_and_print_txt(cnn_out.cipher(), decryptor, encoder, 1<<logn, 256, 2, output); cnn_out.print_parms();
	cout << "remaining level : " << cnn_out.cipher().chain_index() << endl;
	cout << "scale: " << cnn_out.cipher().scale() << endl
		 << endl;
	output << "remaining level : " << cnn_out.cipher().chain_index() << endl;
	output << "scale: " << cnn_out.cipher().scale() << endl
		   << endl;
}
// void multiplexed_parallel_convolution_phantom(const TensorCipher &cnn_in, TensorCipher &cnn_out, int co, int st, int fh, int fw, const vector<double> &data, vector<double> running_var, vector<double> constant_weight, double epsilon, CKKSEncoder &encoder, Encryptor &encryptor, Evaluator &evaluator, GaloisKeys &gal_keys, vector<PhantomCiphertext> &cipher_pool, bool end)
void multiplexed_parallel_convolution_phantom(const TensorCipher &cnn_in, TensorCipher &cnn_out, int co, int st, int fh, int fw, const vector<double> &data, vector<double> running_var, vector<double> constant_weight, double epsilon, CKKSEvaluator &ckksevaluator, vector<PhantomCiphertext> &cipher_pool, bool end)
{
	// set parameters
	vector<double> conv_data;
	int ki = cnn_in.k(), hi = cnn_in.h(), wi = cnn_in.w(), ci = cnn_in.c(), ti = cnn_in.t(), pi = cnn_in.p(), logn = cnn_in.logn();
	int ko = 0, ho = 0, wo = 0, to = 0, po = 0;

	// error check
	if (st != 1 && st != 2)
		throw invalid_argument("supported st is only 1 or 2"); // check if st is 1 or 2
	if (static_cast<int>(data.size()) != fh * fw * ci * co)
		throw std::invalid_argument("the size of data vector is not ker x ker x h x h"); // check if the size of data vector is kernel x kernel x h x h'
	if (log2_long(ki) == -1)
		throw std::invalid_argument("ki is not power of two");

	if (static_cast<int>(running_var.size()) != co || static_cast<int>(constant_weight.size()) != co)
		throw std::invalid_argument("the size of running_var or weight is not correct");
	for (auto num : running_var)
		if (num < pow(10, -16) && num > -pow(10, -16))
			throw std::invalid_argument("the size of running_var is too small. nearly zero.");

	// set ho, wo, ko
	if (st == 1)
	{
		ho = hi;
		wo = wi;
		ko = ki;
	}
	else if (st == 2)
	{
		if (hi % 2 == 1 || wi % 2 == 1)
			throw std::invalid_argument("hi or wi is not even");
		ho = hi / 2;
		wo = wi / 2;
		ko = 2 * ki;
	}

	// set to, po, q
	long n = 1 << logn;
	to = (co + ko * ko - 1) / (ko * ko);
	po = pow2(floor_to_int(log(static_cast<double>(n) / static_cast<double>(ko * ko * ho * wo * to)) / log(2.0)));
	long q = (co + pi - 1) / pi;

	// check if pi, po | n
	if (n % pi != 0)
		throw std::out_of_range("n is not divisible by pi");
	if (n % po != 0)
		throw std::out_of_range("n is not divisible by po");

	// check if ki^2 hi wi ti pi <= n and ko^2 ho wo to po <= n
	if (ki * ki * hi * wi * ti * pi > n)
		throw std::out_of_range("ki^2 hi wi ti pi is larger than n");
	if (ko * ko * ho * wo * to * po > (1 << logn))
		throw std::out_of_range("ko^2 ho wo to po is larger than n");

	// variable
	vector<vector<vector<vector<double>>>> weight(fh, vector<vector<vector<double>>>(fw, vector<vector<double>>(ci, vector<double>(co, 0.0))));			  // weight tensor
	vector<vector<vector<vector<double>>>> compact_weight_vec(fh, vector<vector<vector<double>>>(fw, vector<vector<double>>(q, vector<double>(n, 0.0)))); // multiplexed parallel shifted weight tensor
	vector<vector<vector<vector<double>>>> select_one(co, vector<vector<vector<double>>>(ko * ho, vector<vector<double>>(ko * wo, vector<double>(to, 0.0))));
	vector<vector<double>> select_one_vec(co, vector<double>(1 << logn, 0.0));

	// weight setting
	for (int i1 = 0; i1 < fh; i1++)
	{
		for (int i2 = 0; i2 < fw; i2++)
		{
			for (int j3 = 0; j3 < ci; j3++)
			{
				for (int j4 = 0; j4 < co; j4++)
				{
					weight[i1][i2][j3][j4] = data[fh * fw * ci * j4 + fh * fw * j3 + fw * i1 + i2];
				}
			}
		}
	}

	// compact shifted weight vector setting
	for (int i1 = 0; i1 < fh; i1++)
	{
		for (int i2 = 0; i2 < fw; i2++)
		{
			for (int i9 = 0; i9 < q; i9++)
			{
				for (int j8 = 0; j8 < n; j8++)
				{
					int j5 = ((j8 % (n / pi)) % (ki * ki * hi * wi)) / (ki * wi), j6 = (j8 % (n / pi)) % (ki * wi), i7 = (j8 % (n / pi)) / (ki * ki * hi * wi), i8 = j8 / (n / pi);
					if (j8 % (n / pi) >= ki * ki * hi * wi * ti || i8 + pi * i9 >= co || ki * ki * i7 + ki * (j5 % ki) + j6 % ki >= ci || (j6 / ki) - (fw - 1) / 2 + i2 < 0 || (j6 / ki) - (fw - 1) / 2 + i2 > wi - 1 || (j5 / ki) - (fh - 1) / 2 + i1 < 0 || (j5 / ki) - (fh - 1) / 2 + i1 > hi - 1)
						compact_weight_vec[i1][i2][i9][j8] = 0.0;
					else
					{
						compact_weight_vec[i1][i2][i9][j8] = weight[i1][i2][ki * ki * i7 + ki * (j5 % ki) + j6 % ki][i8 + pi * i9];
					}
				}
			}
		}
	}

	// select one setting
	for (int j4 = 0; j4 < co; j4++)
	{
		for (int v1 = 0; v1 < ko * ho; v1++)
		{
			for (int v2 = 0; v2 < ko * wo; v2++)
			{
				for (int u3 = 0; u3 < to; u3++)
				{
					if (ko * ko * u3 + ko * (v1 % ko) + v2 % ko == j4)
						select_one[j4][v1][v2][u3] = constant_weight[j4] / sqrt(running_var[j4] + epsilon);
					else
						select_one[j4][v1][v2][u3] = 0.0;
				}
			}
		}
	}

	// select one vector setting
	for (int j4 = 0; j4 < co; j4++)
	{
		for (int v1 = 0; v1 < ko * ho; v1++)
		{
			for (int v2 = 0; v2 < ko * wo; v2++)
			{
				for (int u3 = 0; u3 < to; u3++)
				{
					select_one_vec[j4][ko * ko * ho * wo * u3 + ko * wo * v1 + v2] = select_one[j4][v1][v2][u3];
				}
			}
		}
	}

	// PhantomCiphertext variables
	PhantomCiphertext *ctxt_in = &cipher_pool[0], *ct_zero = &cipher_pool[1], *temp = &cipher_pool[2], *sum = &cipher_pool[3], *total_sum = &cipher_pool[4], *var = &cipher_pool[5];

	// PhantomCiphertext input
	*ctxt_in = cnn_in.cipher();

	// rotated input precomputation
	vector<vector<PhantomCiphertext *>> ctxt_rot(fh, vector<PhantomCiphertext *>(fw));
	// if(fh != 3 || fw != 3) throw std::invalid_argument("fh and fw should be 3");
	if (fh % 2 == 0 || fw % 2 == 0)
		throw std::invalid_argument("fh and fw should be odd");
	for (int i1 = 0; i1 < fh; i1++)
	{
		for (int i2 = 0; i2 < fw; i2++)
		{
			if (i1 == (fh - 1) / 2 && i2 == (fw - 1) / 2)
				ctxt_rot[i1][i2] = ctxt_in; // i1=(fh-1)/2, i2=(fw-1)/2 means ctxt_in
			else if ((i1 == (fh - 1) / 2 && i2 > (fw - 1) / 2) || i1 > (fh - 1) / 2)
				ctxt_rot[i1][i2] = &cipher_pool[6 + fw * i1 + i2 - 1];
			else
				ctxt_rot[i1][i2] = &cipher_pool[6 + fw * i1 + i2];
		}
	}
	// ctxt_rot[0][0] = &cipher_pool[6];	ctxt_rot[0][1] = &cipher_pool[7];	ctxt_rot[0][2] = &cipher_pool[8];
	// ctxt_rot[1][0] = &cipher_pool[9];	ctxt_rot[1][1] = ctxt_in;			ctxt_rot[1][2] = &cipher_pool[10];		// i1=1, i2=1 means ctxt_in
	// ctxt_rot[2][0] = &cipher_pool[11];	ctxt_rot[2][1] = &cipher_pool[12];	ctxt_rot[2][2] = &cipher_pool[13];
	for (int i1 = 0; i1 < fh; i1++)
	{
		for (int i2 = 0; i2 < fw; i2++)
		{
			*ctxt_rot[i1][i2] = *ctxt_in;
			memory_save_rotate(*ctxt_rot[i1][i2], *ctxt_rot[i1][i2], ki * ki * wi * (i1 - (fh - 1) / 2) + ki * (i2 - (fw - 1) / 2), ckksevaluator);
		}
	}

	// generate zero PhantomCiphertext
	vector<double> zero(1 << logn, 0.0);
	PhantomPlaintext plain;
	ckksevaluator.encoder.encode(zero, ctxt_in->scale(), plain);
	ckksevaluator.encryptor.encrypt(plain, *ct_zero); // ct_zero: original scaling factor

	for (int i9 = 0; i9 < q; i9++)
	{
		// weight multiplication
		// cout << "multiplication by filter coefficients" << endl;
		for (int i1 = 0; i1 < fh; i1++)
		{
			for (int i2 = 0; i2 < fw; i2++)
			{
				// *temp = *ctxt_in;
				// memory_save_rotate(*temp, *temp, k*k*l*(i1-(kernel-1)/2) + k*(i2-(kernel-1)/2), scale_evaluator, gal_keys);
				// scale_evaluator.multiply_vector_inplace_scaleinv(*temp, compact_weight_vec[i1][i2][i9]);		// temp: double scaling factor
				// evaluator.multiply_vector_reduced_error(*ctxt_rot[i1][i2], compact_weight_vec[i1][i2][i9], *temp);		// temp: double scaling factor
				ckksevaluator.evaluator.multiply_vector_reduced_error(*ctxt_rot[i1][i2], compact_weight_vec[i1][i2][i9], *temp);
				if (i1 == 0 && i2 == 0)
					*sum = *temp; // sum: double scaling factor
				else
					ckksevaluator.evaluator.add_inplace_reduced_error(*sum, *temp);
			}
		}
		ckksevaluator.evaluator.rescale_to_next_inplace(*sum);
		*var = *sum;

		// summation for all input channels
		// cout << "summation for all input channels" << endl;
		int d = log2_long(ki), c = log2_long(ti);
		for (int x = 0; x < d; x++)
		{
			*temp = *var;
			//	scale_evaluator.rotate_vector(temp, pow2(x), gal_keys, temp);
			memory_save_rotate(*temp, *temp, pow2(x), ckksevaluator);
			ckksevaluator.evaluator.add_inplace_reduced_error(*var, *temp);
		}
		for (int x = 0; x < d; x++)
		{
			*temp = *var;
			//	scale_evaluator.rotate_vector(temp, pow2(x)*k*l, gal_keys, temp);
			memory_save_rotate(*temp, *temp, pow2(x) * ki * wi, ckksevaluator);
			ckksevaluator.evaluator.add_inplace_reduced_error(*var, *temp);
		}
		if (c == -1)
		{
			*sum = *ct_zero;
			for (int x = 0; x < ti; x++)
			{
				*temp = *var;
				//	scale_evaluator.rotate_vector(temp, k*k*l*l*x, gal_keys, temp);
				memory_save_rotate(*temp, *temp, ki * ki * hi * wi * x, ckksevaluator);
				ckksevaluator.evaluator.add_inplace_reduced_error(*sum, *temp);
			}
			*var = *sum;
		}
		else
		{
			for (int x = 0; x < c; x++)
			{
				*temp = *var;
				//	scale_evaluator.rotate_vector(temp, pow2(x)*k*k*l*l, gal_keys, temp);
				memory_save_rotate(*temp, *temp, pow2(x) * ki * ki * hi * wi, ckksevaluator);
				ckksevaluator.evaluator.add_inplace_reduced_error(*var, *temp);
			}
		}

		// collecting valid values into one PhantomCiphertext.
		// cout << "collecting valid values into one PhantomCiphertext." << endl;
		for (int i8 = 0; i8 < pi && pi * i9 + i8 < co; i8++)
		{
			int j4 = pi * i9 + i8;
			if (j4 >= co)
				throw std::out_of_range("the value of j4 is out of range!");

			*temp = *var;
			memory_save_rotate(*temp, *temp, (n / pi) * (j4 % pi) - j4 % ko - (j4 / (ko * ko)) * ko * ko * ho * wo - ((j4 % (ko * ko)) / ko) * ko * wo, ckksevaluator);
			ckksevaluator.evaluator.multiply_vector_inplace_reduced_error(*temp, select_one_vec[j4]); // temp: double scaling factor
			if (i8 == 0 && i9 == 0)
				*total_sum = *temp; // total_sum: double scaling factor
			else
				ckksevaluator.evaluator.add_inplace_reduced_error(*total_sum, *temp);
		}
	}
	ckksevaluator.evaluator.rescale_to_next_inplace(*total_sum);
	*var = *total_sum;

	// po copies
	if (end == false)
	{
		// cout << "po copies" << endl;
		*sum = *ct_zero;
		for (int u6 = 0; u6 < po; u6++)
		{
			*temp = *var;
			memory_save_rotate(*temp, *temp, -u6 * (n / po), ckksevaluator);
			ckksevaluator.evaluator.add_inplace_reduced_error(*sum, *temp); // sum: original scaling factor.
		}
		*var = *sum;
	}

	cnn_out = TensorCipher(logn, ko, ho, wo, co, to, po, *var);
}
// void multiplexed_parallel_batch_norm_phantom(const TensorCipher &cnn_in, TensorCipher &cnn_out, vector<double> bias, vector<double> running_mean, vector<double> running_var, vector<double> weight, double epsilon, CKKSEncoder &encoder, Encryptor &encryptor, Evaluator &evaluator, double B, bool end)
void multiplexed_parallel_batch_norm_phantom(const TensorCipher &cnn_in, TensorCipher &cnn_out, vector<double> bias, vector<double> running_mean, vector<double> running_var, vector<double> weight, double epsilon, CKKSEvaluator &ckksevaluator, double B, bool end)
{
	// parameter setting
	int ki = cnn_in.k(), hi = cnn_in.h(), wi = cnn_in.w(), ci = cnn_in.c(), ti = cnn_in.t(), pi = cnn_in.p(), logn = cnn_in.logn();
	int ko = ki, ho = hi, wo = wi, co = ci, to = ti, po = pi;

	// error check
	if (static_cast<int>(bias.size()) != ci || static_cast<int>(running_mean.size()) != ci || static_cast<int>(running_var.size()) != ci || static_cast<int>(weight.size()) != ci)
		throw std::invalid_argument("the size of bias, running_mean, running_var, or weight are not correct");
	for (auto num : running_var)
		if (num < pow(10, -16) && num > -pow(10, -16))
			throw std::invalid_argument("the size of running_var is too small. nearly zero.");
	if (hi * wi * ci > 1 << logn)
		throw std::invalid_argument("hi*wi*ci should not be larger than n");

	// generate g vector
	vector<double> g(1 << logn, 0.0);

	// set f value
	long n = 1 << logn;

	// check if pi | n
	if (n % pi != 0)
		throw std::out_of_range("n is not divisible by pi");

	// set g vector
	for (int v4 = 0; v4 < n; v4++)
	{
		int v1 = ((v4 % (n / pi)) % (ki * ki * hi * wi)) / (ki * wi), v2 = (v4 % (n / pi)) % (ki * wi), u3 = (v4 % (n / pi)) / (ki * ki * hi * wi);
		if (ki * ki * u3 + ki * (v1 % ki) + v2 % ki >= ci || v4 % (n / pi) >= ki * ki * hi * wi * ti)
			g[v4] = 0.0;
		else
		{
			int idx = ki * ki * u3 + ki * (v1 % ki) + v2 % ki;
			g[v4] = (running_mean[idx] * weight[idx] / sqrt(running_var[idx] + epsilon) - bias[idx]) / B;
		}
	}

	// encode & encrypt
	PhantomPlaintext plain;
	PhantomCiphertext cipher_g;
	PhantomCiphertext temp;
	temp = cnn_in.cipher();
	ckksevaluator.encoder.encode(g, temp.scale(), plain);
	ckksevaluator.encryptor.encrypt(plain, cipher_g);

	// batch norm
	ckksevaluator.evaluator.sub_inplace_reduced_error(temp, cipher_g);

	cnn_out = TensorCipher(logn, ko, ho, wo, co, to, po, temp);
}
// void ReLU_phantom(const TensorCipher &cnn_in, TensorCipher &cnn_out, long comp_no, vector<int> deg, long alpha, vector<Tree> &tree, double scaled_val, long scalingfactor, Encryptor &encryptor, Evaluator &evaluator, Decryptor &decryptor, CKKSEncoder &encoder, PublicKey &public_key, SecretKey &secret_key, RelinKeys &relin_keys, double scale)
void ReLU_phantom(const TensorCipher &cnn_in, TensorCipher &cnn_out, long comp_no, vector<int> deg, long alpha, vector<Tree> &tree, double scaled_val, long scalingfactor, CKKSEvaluator &ckksevaluator, double scale)
{
	// parameter setting
	int ki = cnn_in.k(), hi = cnn_in.h(), wi = cnn_in.w(), ci = cnn_in.c(), ti = cnn_in.t(), pi = cnn_in.p(), logn = cnn_in.logn();
	int ko = ki, ho = hi, wo = wi, co = ci, to = ti, po = pi;

	// error check
	if (hi * wi * ci > 1 << logn)
		throw std::invalid_argument("hi*wi*ci should not be larger than n");

	// ReLU
	PhantomCiphertext temp;
	temp = cnn_in.cipher();
	minimax_ReLU_phantom(comp_no, deg, alpha, tree, scaled_val, scalingfactor, ckksevaluator, temp, temp);

	cnn_out = TensorCipher(logn, ko, ho, wo, co, to, po, temp);
}
void cipher_add_phantom_print(const TensorCipher &cnn1, const TensorCipher &cnn2, TensorCipher &destination, CKKSEvaluator &ckksevaluator, ofstream &output)
{
	cout << "cipher add..." << endl;
	output << "cipher add..." << endl;
	int logn = cnn1.logn();
	cnn_add_phantom(cnn1, cnn2, destination, ckksevaluator);
	// cout << "cipher add result" << endl;
	auto decry_and_print_temp = destination.cipher();
	decrypt_and_print(decry_and_print_temp, ckksevaluator, 1 << logn, 256, 2);
	destination.print_parms();
	// decrypt_and_print_txt(cnn_out.cipher(), decryptor, encoder, 1<<logn, 256, 2, output); cnn_out.print_parms();
	cout << "remaining level : " << destination.cipher().chain_index() << endl;
	cout << "scale: " << destination.cipher().scale() << endl
		 << endl;
	output << "remaining level : " << destination.cipher().chain_index() << endl;
	output << "scale: " << destination.cipher().scale() << endl
		   << endl;
}
void cnn_add_phantom(const TensorCipher &cnn1, const TensorCipher &cnn2, TensorCipher &destination, CKKSEvaluator &ckksevaluator)
{
	// parameter setting
	int k1 = cnn1.k(), h1 = cnn1.h(), w1 = cnn1.w(), c1 = cnn1.c(), t1 = cnn1.t(), p1 = cnn1.p(), logn1 = cnn1.logn();
	int k2 = cnn2.k(), h2 = cnn2.h(), w2 = cnn2.w(), c2 = cnn2.c(), t2 = cnn2.t(), p2 = cnn2.p(), logn2 = cnn2.logn();

	// error check
	if (k1 != k2 || h1 != h2 || w1 != w2 || c1 != c2 || t1 != t2 || p1 != p2 || logn1 != logn2)
		throw std::invalid_argument("the parameters of cnn1 and cnn2 are not the same");

	// addition
	PhantomCiphertext temp1, temp2;
	temp1 = cnn1.cipher();
	temp2 = cnn2.cipher();
	ckksevaluator.evaluator.add_inplace_reduced_error(temp1, temp2);

	destination = TensorCipher(logn1, k1, h1, w1, c1, t1, p1, temp1);
}
void multiplexed_parallel_downsampling_phantom(const TensorCipher &cnn_in, TensorCipher &cnn_out, CKKSEvaluator &ckksevaluator)
{
	// parameter setting
	int ki = cnn_in.k(), hi = cnn_in.h(), wi = cnn_in.w(), ci = cnn_in.c(), ti = cnn_in.t(), pi = cnn_in.p(), logn = cnn_in.logn();
	int ko = 0, ho = 0, wo = 0, co = 0, to = 0, po = 0;

	// parameter setting
	long n = 1 << logn;
	ko = 2 * ki;
	ho = hi / 2;
	wo = wi / 2;
	to = ti / 2;
	co = 2 * ci;
	po = pow2(floor_to_int(log(static_cast<double>(n) / static_cast<double>(ko * ko * ho * wo * to)) / log(2.0)));

	// error check
	if (ti % 8 != 0)
		throw std::invalid_argument("ti is not multiple of 8");
	if (hi % 2 != 0)
		throw std::invalid_argument("hi is not even");
	if (wi % 2 != 0)
		throw std::invalid_argument("wi is not even");
	if (n % po != 0)
		throw std::out_of_range("n is not divisible by po"); // check if po | n

	// variables
	vector<vector<vector<double>>> select_one_vec(ki, vector<vector<double>>(ti, vector<double>(1 << logn, 0.0)));
	PhantomCiphertext ct, sum, temp;
	ct = cnn_in.cipher();

	// selecting tensor vector setting
	for (int w1 = 0; w1 < ki; w1++)
	{
		for (int w2 = 0; w2 < ti; w2++)
		{
			for (int v4 = 0; v4 < 1 << logn; v4++)
			{
				int j5 = (v4 % (ki * ki * hi * wi)) / (ki * wi), j6 = v4 % (ki * wi), i7 = v4 / (ki * ki * hi * wi);
				if (v4 < ki * ki * hi * wi * ti && (j5 / ki) % 2 == 0 && (j6 / ki) % 2 == 0 && (j5 % ki) == w1 && i7 == w2)
					select_one_vec[w1][w2][v4] = 1.0;
				else
					select_one_vec[w1][w2][v4] = 0.0;
			}
		}
	}

	for (int w1 = 0; w1 < ki; w1++)
	{
		for (int w2 = 0; w2 < ti; w2++)
		{
			temp = ct;
			ckksevaluator.evaluator.multiply_vector_inplace_reduced_error(temp, select_one_vec[w1][w2]);

			int w3 = ((ki * w2 + w1) % (2 * ko)) / 2, w4 = (ki * w2 + w1) % 2, w5 = (ki * w2 + w1) / (2 * ko);
			memory_save_rotate(temp, temp, ki * ki * hi * wi * w2 + ki * wi * w1 - ko * ko * ho * wo * w5 - ko * wo * w3 - ki * w4 - ko * ko * ho * wo * (ti / 8), ckksevaluator);
			if (w1 == 0 && w2 == 0)
				sum = temp;
			else
				ckksevaluator.evaluator.add_inplace_reduced_error(sum, temp);
		}
	}
	ckksevaluator.evaluator.rescale_to_next_inplace(sum); // added
	ct = sum;

	// for fprime packing
	sum = ct;
	for (int u6 = 1; u6 < po; u6++)
	{
		temp = ct;
		memory_save_rotate(temp, temp, -(n / po) * u6, ckksevaluator);
		ckksevaluator.evaluator.add_inplace_reduced_error(sum, temp);
	}
	ct = sum;

	cnn_out = TensorCipher(logn, ko, ho, wo, co, to, po, ct);
}

void averagepooling_phantom_scale(const TensorCipher &cnn_in, TensorCipher &cnn_out, CKKSEvaluator &ckksevaluator, double B, ofstream &output)
{
	// parameter setting
	int ki = cnn_in.k(), hi = cnn_in.h(), wi = cnn_in.w(), ci = cnn_in.c(), ti = cnn_in.t(), pi = cnn_in.p(), logn = cnn_in.logn();
	int ko = 1, ho = 1, wo = 1, co = ci, to = ti;

	if (log2_long(hi) == -1)
		throw std::invalid_argument("hi is not power of two");
	if (log2_long(wi) == -1)
		throw std::invalid_argument("wi is not power of two");

	PhantomCiphertext ct, temp, sum;
	ct = cnn_in.cipher();

	// sum_hiwi
	// cout << "sum hiwi" << endl;

	for (int x = 0; x < log2_long(wi); x++)
	{
		temp = ct;
		//	scale_evaluator.rotate_vector_inplace(temp, pow2(x)*k, gal_keys);
		memory_save_rotate(temp, temp, pow2(x) * ki, ckksevaluator);
		ckksevaluator.evaluator.add_inplace_reduced_error(ct, temp);
	}
	for (int x = 0; x < log2_long(hi); x++)
	{
		temp = ct;
		//	scale_evaluator.rotate_vector_inplace(temp, pow2(x)*k*k*l, gal_keys);
		memory_save_rotate(temp, temp, pow2(x) * ki * ki * wi, ckksevaluator);
		ckksevaluator.evaluator.add_inplace_reduced_error(ct, temp);
	}

	// cout << "sum l^2 results" << endl;
	// output << "sum l^2 results" << endl;
	// decrypt_and_print_txt(ct, decryptor, encoder, 1<<logn, 256, 2, output);

	// final
	// cout << "final" << endl;
	vector<double> select_one(1 << logn, 0.0), zero(1 << logn, 0.0);
	for (int s = 0; s < ki; s++)
	{
		for (int u = 0; u < ti; u++)
		{
			int p = ki * u + s;
			temp = ct;
			//	scale_evaluator.rotate_vector_inplace(temp, -p*k + k*k*l*l*u + k*l*s, gal_keys);
			memory_save_rotate(temp, temp, -p * ki + ki * ki * hi * wi * u + ki * wi * s,ckksevaluator);
			select_one = zero;
			// for(int i=0; i<k; i++) select_one[(k*u+s)*k+i] = 1.0 / static_cast<double>(l*l);
			for (int i = 0; i < ki; i++)
				select_one[(ki * u + s) * ki + i] = B / static_cast<double>(hi * wi);

				ckksevaluator.evaluator.multiply_vector_inplace_reduced_error(temp, select_one);
			if (u == 0 && s == 0)
				sum = temp; // double scaling factor
			else
			ckksevaluator.evaluator.add_inplace_reduced_error(sum, temp);
		}

		// cout << "final iteration results" << endl;
		// output << "final iteration results" << endl;
		// decrypt_and_print_txt(sum, decryptor, encoder, 1<<logn, 256, 2, output);
	}
	ckksevaluator.evaluator.rescale_to_next_inplace(sum);

	// cout << "rescaling results" << endl;
	// output << "rescaling results" << endl;
	// decrypt_and_print_txt(sum, decryptor, encoder, 1<<logn, 256, 2, output);

	cnn_out = TensorCipher(logn, ko, ho, wo, co, to, 1, sum);
}
void matrix_multiplication_phantom(const TensorCipher &cnn_in, TensorCipher &cnn_out, vector<double> matrix, vector<double> bias, int q, int r, CKKSEvaluator &ckksevaluator)
{
	// parameter setting
	int ki = cnn_in.k(), hi = cnn_in.h(), wi = cnn_in.w(), ci = cnn_in.c(), ti = cnn_in.t(), pi = cnn_in.p(), logn = cnn_in.logn();
	int ko = ki, ho = hi, wo = wi, co = ci, to = ti, po = pi;

	if (static_cast<int>(matrix.size()) != q * r)
		throw std::invalid_argument("the size of matrix is not q*r");
	if (static_cast<int>(bias.size()) != q)
		throw std::invalid_argument("the size of bias is not q");

	// generate matrix and bias
	vector<vector<double>> W(q + r - 1, vector<double>(1 << logn, 0.0));
	vector<double> b(1 << logn, 0.0);

	for (int z = 0; z < q; z++)
		b[z] = bias[z];
	for (int i = 0; i < q; i++)
	{
		for (int j = 0; j < r; j++)
		{
			W[i - j + r - 1][i] = matrix[i * r + j];
			if (i - j + r - 1 < 0 || i - j + r - 1 >= q + r - 1)
				throw std::out_of_range("i-j+r-1 is out of range");
			if (i * r + j < 0 || i * r + j >= static_cast<int>(matrix.size()))
				throw std::out_of_range("i*r+j is out of range");
		}
	}

	// matrix multiplication
	PhantomCiphertext ct, temp, sum;
	ct = cnn_in.cipher();
	for (int s = 0; s < q + r - 1; s++)
	{
		temp = ct;
		//	scale_evaluator.rotate_vector_inplace(temp, r-1-s, gal_keys);
		memory_save_rotate(temp, temp, r - 1 - s, ckksevaluator);
		ckksevaluator.evaluator.multiply_vector_inplace_reduced_error(temp, W[s]);
		if (s == 0)
			sum = temp;
		else
		ckksevaluator.evaluator.add_inplace_reduced_error(sum, temp);
	}
	ckksevaluator.evaluator.rescale_to_next_inplace(sum);

	cnn_out = TensorCipher(logn, ko, ho, wo, co, to, po, sum);
}
void minimax_ReLU_phantom(long comp_no, vector<int> deg, long alpha, vector<Tree> &tree, double scaled_val, long scalingfactor, CKKSEvaluator &ckksevaluator, PhantomCiphertext &cipher_in, PhantomCiphertext &cipher_res)
{
	// variables
	vector<vector<double>> decomp_coeff(comp_no, vector<double>(0));
	vector<double> scale_val(comp_no, 0.0);
	PhantomPlaintext plain_half;
	PhantomCiphertext cipher_temp, cipher_half, cipher_x;

	// ifstream and scale
	string str;
	string addr = "../result";
	str = addr + "/d" + to_string(alpha) + ".txt";
	ifstream in(str);

	// scaled value setting
	scale_val[0] = 1.0;
	for (int i = 1; i < comp_no; i++)
		scale_val[i] = 2.0;
	scale_val[comp_no - 1] = scaled_val;

	// print degrees and coefficients of the component polynomials of minimax composite polynomial
	// for(int i=0; i<comp_no; i++) cout << deg[i] << " ";
	// cout << endl;
	for (int i = 0; i < comp_no; i++)
	{
		for (int j = 0; j < coeff_number(deg[i], tree[i]); j++)
		{
			double temp;
			in >> temp;
			decomp_coeff[i].emplace_back(temp);
			// cout << decomp_coeff[i][j] << " ";
		}
		// cout << endl;
	}

	// scale coefficients properly so that unnecessary level consumptions do not occur
	for (int i = 0; i < comp_no - 1; i++)
		for (int j = 0; j < coeff_number(deg[i], tree[i]); j++)
			decomp_coeff[i][j] /= scale_val[i + 1];
	for (int j = 0; j < coeff_number(deg[comp_no - 1], tree[comp_no - 1]); j++)
		decomp_coeff[comp_no - 1][j] *= 0.5; // scale

	// generation of half PhantomCiphertext
	long n = cipher_in.poly_modulus_degree() / 2;
	vector<double> m_half(n);
	for (int i = 0; i < n; i++)
		m_half[i] = 0.5;
	cipher_x = cipher_in;

	// evaluating pk ... p1(x) / 2
	for (int i = 0; i < comp_no; ++i)
	{
		// cout << "*******************************************" << endl;
		// cout << "               No: " << i << endl;
		eval_polynomial_integrate(ckksevaluator, cipher_x, cipher_x, deg[i], decomp_coeff[i], tree[i]);
		decrypt_and_print_part(cipher_x, ckksevaluator, n, 0, 5);
	}

	// x(1+sgn(x))/2 from sgn(x)/2
	ckksevaluator.encoder.encode(m_half, cipher_x.scale(), plain_half);
	ckksevaluator.encryptor.encrypt(plain_half, cipher_half);
	ckksevaluator.evaluator.add_reduced_error(cipher_x, cipher_half, cipher_temp);
	ckksevaluator.evaluator.multiply_reduced_error(cipher_temp, cipher_in, *(ckksevaluator.relin_keys), cipher_res);
	ckksevaluator.evaluator.rescale_to_next_inplace(cipher_res);
}

void memory_save_rotate(const PhantomCiphertext &cipher_in, PhantomCiphertext &cipher_out, int steps, CKKSEvaluator &ckksevaluator)
{
	long n = cipher_in.poly_modulus_degree() / 2;
	PhantomCiphertext temp = cipher_in;
	steps = (steps + n) % n; // 0 ~ n-1
	int first_step = 0;

	if (34 <= steps && steps <= 55)
		first_step = 33;
	else if (57 <= steps && steps <= 61)
		first_step = 33;
	else
		first_step = 0;
	if (steps == 0)
		return; // no rotation

	if (first_step == 0)
		ckksevaluator.evaluator.rotate_vector_inplace(temp, steps, *(ckksevaluator.galois_keys));
	else
	{
		// evaluator.rotate_vector_inplace(temp, first_step, gal_keys);
		ckksevaluator.evaluator.rotate_vector_inplace(temp, first_step, *(ckksevaluator.galois_keys));
		// evaluator.rotate_vector_inplace(temp, steps-first_step, gal_keys);
		ckksevaluator.evaluator.rotate_vector_inplace(temp, steps - first_step, *(ckksevaluator.galois_keys));
	}

	cipher_out = temp;
	//	else scale_evaluator.rotate_vector(cipher_in, steps, gal_keys, cipher_out);
}

void MultipleAdd_SEAL(CKKSEvaluator &ckksevaluator, PhantomCiphertext &cipher, PhantomCiphertext &result, long long n)
{
	long long k, abs_n;
	long long *binary;
	//	PhantomCiphertext temp;
	if (n >= 0)
		abs_n = n;
	else
		abs_n = -n;

	for (k = 1; k < 100; k++)
	{
		if (abs_n < pow2(k))
			break;
	}

	binary = new long long[k];
	for (long i = 0; i < k; i++)
	{
		binary[i] = (abs_n / pow2(i)) % 2;
	}

	// evaluator.add(cipher, cipher, result);
	ckksevaluator.evaluator.add(cipher, cipher, result);
	if (binary[k - 2] == 1)
		ckksevaluator.evaluator.add_inplace(result, cipher);

	for (long i = k - 3; i >= 0; i--)
	{
		ckksevaluator.evaluator.add_inplace(result, result);
		if (binary[i] == 1)
			ckksevaluator.evaluator.add_inplace(result, cipher);
	}

	if (n < 0)
		ckksevaluator.evaluator.negate_inplace(result);
}
void test_evaluation(CKKSEvaluator &ckksevaluator, const PhantomCiphertext &cipher_in, PhantomCiphertext &cipher_out);
void geneT0T1(CKKSEvaluator &ckksevaluator, PhantomCiphertext &T0, PhantomCiphertext &T1, PhantomCiphertext &cipher)
{

	double scale = cipher.scale();
	long n = cipher.poly_modulus_degree() / 2;
	//	vector<double> m_one(n), m_scaled(n);
	vector<double> m_one(n);

	// ctxt_1
	for (int i = 0; i < n; i++)
		m_one[i] = 1.0;
	PhantomPlaintext plain_1;
	ckksevaluator.encoder.encode(m_one, scale, plain_1);
	PhantomCiphertext ctxt_1;
	ckksevaluator.encryptor.encrypt(plain_1, ctxt_1);

	T0 = ctxt_1;
	T1 = cipher;
}
void evalT(CKKSEvaluator &ckksevaluator, PhantomCiphertext &Tmplusn, const PhantomCiphertext &Tm, const PhantomCiphertext &Tn, const PhantomCiphertext &Tmminusn)
{
	PhantomCiphertext temp;
	ckksevaluator.evaluator.multiply_reduced_error(Tm, Tn, *(ckksevaluator.relin_keys), temp);
	ckksevaluator.evaluator.add_inplace_reduced_error(temp, temp);
	ckksevaluator.evaluator.rescale_to_next_inplace(temp);
	ckksevaluator.evaluator.sub_reduced_error(temp, Tmminusn, Tmplusn);
}
void eval_polynomial_integrate(CKKSEvaluator &ckksevaluator, PhantomCiphertext &res, PhantomCiphertext &cipher, long deg, const vector<double> &decomp_coeff, Tree &tree)
{
	double scale = cipher.scale(); // ex) 2^42. exact value.
	long n = cipher.poly_modulus_degree() / 2;
	long total_depth = ceil_to_int(log(static_cast<double>(deg + 1)) / log(2.0)); // required minimum depth considering both scalar and nonscalar multiplications
	PhantomCiphertext temp1, temp2, state, ctxt_zero;
	evaltype eval_type = tree.type;
	vector<long> decomp_deg(pow2(tree.depth + 1), -1);
	vector<long> start_index(pow2(tree.depth + 1), -1);
	vector<std::unique_ptr<PhantomCiphertext>> T(100);
	vector<std::unique_ptr<PhantomCiphertext>> pt(100);
	for (size_t i = 0; i < 100; i++)
		T[i] = nullptr;
	for (size_t i = 0; i < 100; i++)
		pt[i] = nullptr;
	T[0] = std::make_unique<PhantomCiphertext>();
	T[1] = std::make_unique<PhantomCiphertext>();

	// generation of zero PhantomCiphertext
	vector<double> m_coeff(n), m_zero(n, 0.0);
	PhantomPlaintext plain_coeff, plain_zero;
	ckksevaluator.encoder.encode(m_zero, scale * scale, plain_zero); // scaling factor: scale^2 for lazy scaling
	ckksevaluator.encryptor.encrypt(plain_zero, ctxt_zero);

	// set start temp_index
	long num = 0, temp_index;
	if (eval_type == evaltype::oddbaby)
		temp_index = 1;
	else if (eval_type == evaltype::baby)
		temp_index = 0;

	// evaluate decompose polynomial degrees
	decomp_deg[1] = deg;
	for (int i = 1; i <= tree.depth; i++)
	{
		for (int j = pow2(i); j < pow2(i + 1); j++)
		{
			if (j >= static_cast<int>(decomp_deg.size()))
				throw std::invalid_argument("invalid index");
			if (j % 2 == 0)
				decomp_deg[j] = tree.tree[j / 2] - 1;
			else if (j % 2 == 1)
				decomp_deg[j] = decomp_deg[j / 2] - tree.tree[j / 2];
		}
	}

	// compute start index.
	for (int i = 1; i < pow2(tree.depth + 1); i++)
	{
		if (tree.tree[i] == 0)
		{
			start_index[i] = temp_index;
			temp_index += (decomp_deg[i] + 1);
		}
	}

	// generate T0, T1
	geneT0T1(ckksevaluator, *T[0], *T[1], cipher);

	if (eval_type == evaltype::oddbaby)
	{
		// i: depth stage
		for (int i = 1; i <= total_depth; i++)
		{
			// cout << "////////////// stage : " << i << endl;

			// depth i computation. all end points.
			for (int j = 1; j < pow2(tree.depth + 1); j++)
			{
				if (tree.tree[j] == 0 && total_depth + 1 - num_one(j) == i) // depth i stage end points. j: index
				{
					int temp_idx = start_index[j];
					// cout << "pt: " << j << endl;
					pt[j] = std::make_unique<PhantomCiphertext>();
					ckksevaluator.evaluator.multiply_const(*T[1], decomp_coeff[temp_idx], *pt[j]);
					temp_idx += 2;
					for (int k = 3; k <= decomp_deg[j]; k += 2)
					{
						ckksevaluator.evaluator.multiply_const(*T[k], decomp_coeff[temp_idx], temp1);
						ckksevaluator.evaluator.add_inplace_reduced_error(*pt[j], temp1); // this is lazy scaling!!

						temp_idx += 2;
					}
					ckksevaluator.evaluator.rescale_to_next_inplace(*pt[j]);
					// print_cipher(decryptor, encoder, public_key, secret_key, relin_keys, *pt[j]);
					// decrypt_and_print_part(*pt[j], decryptor, encoder, n, 0, 5);
				}
			}

			// depth i computation. all intersection points.
			long inter[40];
			long inter_num = 0;

			for (int j = 1; j < pow2(tree.depth + 1); j++)
			{
				if (tree.tree[j] > 0 && total_depth + 1 - num_one(j) == i && j % 2 == 1) // depth i stage intersection points
				{
					long k = j;
					// cout << "pt: " << j << endl;
					pt[j] = std::make_unique<PhantomCiphertext>();
					ckksevaluator.evaluator.multiply_reduced_error(*T[tree.tree[k]], *pt[2 * k + 1], *(ckksevaluator.relin_keys), *pt[j]);
					k *= 2;
					while (1)
					{
						if (tree.tree[k] == 0)
							break;
						ckksevaluator.evaluator.multiply_reduced_error(*T[tree.tree[k]], *pt[2 * k + 1], *(ckksevaluator.relin_keys), temp1);
						ckksevaluator.evaluator.add_inplace_reduced_error(*pt[j], temp1); // lazy scaling code
						k *= 2;
					}
					ckksevaluator.evaluator.rescale_to_next_inplace(*pt[j]);
					ckksevaluator.evaluator.add_inplace_reduced_error(*pt[j], *pt[k]);
					// print_cipher(decryptor, encoder, public_key, secret_key, relin_keys, *pt[j]);
					// decrypt_and_print_part(*pt[j], decryptor, encoder, n, 0, 5);
				}
			}

			// Ti evaluation
			if (i <= tree.m - 1)
			{
				// cout << "T: " << pow2(i) << endl;
				T[pow2(i)] = std::make_unique<PhantomCiphertext>();
				evalT(ckksevaluator, *T[pow2(i)], *T[pow2(i - 1)], *T[pow2(i - 1)], *T[0]);
				// print_cipher(decryptor, encoder, public_key, secret_key, relin_keys, *T[pow2(i)]);
				// decrypt_and_print_part(*T[pow2(i)], decryptor, encoder, n, 0, 5);
			}

			if (i <= tree.l)
			{
				for (int j = pow2(i - 1) + 1; j <= pow2(i) - 1; j += 2) // T1 is not computed. other odd Tis are computed.
				{
					// cout << "T: " << j << endl;
					T[j] = std::make_unique<PhantomCiphertext>();
					evalT(ckksevaluator, *T[j], *T[pow2(i - 1)], *T[j - pow2(i - 1)], *T[pow2(i) - j]);
					// print_cipher(decryptor, encoder, public_key, secret_key, relin_keys, *T[j]);
					// decrypt_and_print_part(*T[j], decryptor, encoder, n, 0, 5);
				}
			}
		}
		res = *pt[1];
	}
	else if (eval_type == evaltype::baby)
	{
		// i: depth stage
		for (int i = 1; i <= total_depth; i++)
		{
			// cout << "////////////// stage : " << i << endl;

			// depth i computation. all end points.
			for (int j = 1; j < pow2(tree.depth + 1); j++)
			{
				if (tree.tree[j] == 0 && total_depth + 1 - num_one(j) == i) // depth i stage end points. j: index
				{
					int temp_idx = start_index[j];
					// cout << "pt: " << j << endl;
					pt[j] = std::make_unique<PhantomCiphertext>();

					*pt[j] = ctxt_zero;

					for (int k = 0; k <= decomp_deg[j]; k++)
					{
						// cout << "coeff[temp_idx]: " <<  coeff[temp_idx] << endl;
						if (abs(decomp_coeff[temp_idx]) > 1.0 / scale) // to avoid transparent PhantomCiphertext
						{
							if (T[k] == nullptr)
								throw std::runtime_error("T[k] is not set");
							ckksevaluator.evaluator.multiply_const(*T[k], decomp_coeff[temp_idx], temp1);
							ckksevaluator.evaluator.add_inplace(*pt[j], temp1); // this is lazy scaling!!
						}
						temp_idx++;
					}
					ckksevaluator.evaluator.rescale_to_next_inplace(*pt[j]);
					// print_cipher(decryptor, encoder, public_key, secret_key, relin_keys, *pt[j]);
				}
			}

			// depth i computation. all intersection points.
			long inter[40];
			long inter_num = 0;

			for (int j = 1; j < pow2(tree.depth + 1); j++)
			{
				if (tree.tree[j] > 0 && total_depth + 1 - num_one(j) == i) // depth i stage intersection points
				{
					int temp = j;
					bool no_execute = false;
					for (int k = 0; k < inter_num; k++)
					{
						while (1)
						{
							if (temp == inter[k])
							{
								no_execute = true;
								break;
							}
							if (temp % 2 == 0)
								temp /= 2;
							else
								break;
						}
					}

					if (no_execute == false)
					{
						inter[inter_num] = j;
						inter_num += 1;

						long k = j;

						// cout << "pt: " << j << endl;
						pt[j] = std::make_unique<PhantomCiphertext>();
						if (T[tree.tree[k]] == nullptr)
							throw std::runtime_error("T[tree.tree[k]] is not set");
						if (pt[2 * k + 1] == nullptr)
							throw std::runtime_error("pt[2*k+1] is not set");
						ckksevaluator.evaluator.multiply_reduced_error(*T[tree.tree[k]], *pt[2 * k + 1], *(ckksevaluator.relin_keys), *pt[j]);
						k *= 2;

						while (1)
						{
							if (tree.tree[k] == 0)
								break;
							if (T[tree.tree[k]] == nullptr)
								throw std::runtime_error("T[tree.tree[k]] is not set");
							if (pt[2 * k + 1] == nullptr)
								throw std::runtime_error("pt[2*k+1] is not set");
							ckksevaluator.evaluator.multiply_reduced_error(*T[tree.tree[k]], *pt[2 * k + 1], *(ckksevaluator.relin_keys), temp1);
							ckksevaluator.evaluator.add_inplace(*pt[j], temp1); // lazy scaling code
							k *= 2;
						}
						ckksevaluator.evaluator.rescale_to_next_inplace(*pt[j]);
						ckksevaluator.evaluator.add_inplace_reduced_error(*pt[j], *pt[k]);
						// print_cipher(decryptor, encoder, public_key, secret_key, relin_keys, *pt[j]);
					}
				}
			}

			// Ti evaluation
			for (int j = 2; j <= tree.b; j++)
			{
				int g = j;
				if (pow2(i - 1) < g && g <= pow2(i))
				{
					// cout << "T: " << g << endl;
					T[g] = std::make_unique<PhantomCiphertext>();
					if (g % 2 == 0)
					{
						if (T[g / 2] == nullptr)
							throw std::runtime_error("T[g/2] is not set");
						if (T[0] == nullptr)
							throw std::runtime_error("T[0] is not set");
						evalT(ckksevaluator, *T[g], *T[g / 2], *T[g / 2], *T[0]);
					}
					else
					{
						if (T[g / 2] == nullptr)
							throw std::runtime_error("T[g/2] is not set");
						if (T[(g + 1) / 2] == nullptr)
							throw std::runtime_error("T[(g+1)/2] is not set");
						if (T[0] == nullptr)
							throw std::runtime_error("T[0] is not set");
						evalT(ckksevaluator, *T[g], *T[g / 2], *T[(g + 1) / 2], *T[1]);
					}
					// print_cipher(decryptor, encoder, public_key, secret_key, relin_keys, *T[g]);
				}
			}
			for (int j = 1; j <= tree.m - 1; j++)
			{
				int g = pow2(j) * tree.b;
				if (pow2(i - 1) < g && g <= pow2(i))
				{
					// cout << "T: " << g << endl;
					T[g] = std::make_unique<PhantomCiphertext>();
					if (g % 2 == 0)
					{
						if (T[g / 2] == nullptr)
							throw std::runtime_error("T[g/2] is not set");
						if (T[0] == nullptr)
							throw std::runtime_error("T[0] is not set");
						evalT(ckksevaluator, *T[g], *T[g / 2], *T[g / 2], *T[0]);
					}
					else
					{
						if (T[g / 2] == nullptr)
							throw std::runtime_error("T[g/2] is not set");
						if (T[(g + 1) / 2] == nullptr)
							throw std::runtime_error("T[(g+1)/2] is not set");
						if (T[0] == nullptr)
							throw std::runtime_error("T[0] is not set");
						evalT(ckksevaluator, *T[g], *T[g / 2], *T[(g + 1) / 2], *T[1]);
					}
					// print_cipher(decryptor, encoder, public_key, secret_key, relin_keys, *T[g]);
				}
			}
		}
		res = *pt[1];
	}
}
long coeff_number(long deg, Tree &tree)
{
	long num = 0;
	long *decomp_deg = new long[pow2(tree.depth + 1)];
	decomp_deg[1] = deg;
	for (int i = 1; i <= tree.depth; i++)
	{
		for (int j = pow2(i); j < pow2(i + 1); j++)
		{
			if (j % 2 == 0)
				decomp_deg[j] = tree.tree[j / 2] - 1;
			else if (j % 2 == 1)
				decomp_deg[j] = decomp_deg[j / 2] - tree.tree[j / 2];
		}
	}

	for (int i = 0; i < pow2(tree.depth + 1); i++)
	{
		if (tree.tree[i] == 0)
		{
			num += (decomp_deg[i] + 1);
		}
	}
	delete decomp_deg;
	return num;
}
void coeff_change(long comp_no, long deg[], double *coeff[], long type[], vector<Tree> &tree);
long ShowFailure_ReLU(CKKSEvaluator &ckksevaluator, PhantomCiphertext &cipher, vector<double> &x, long precision, long n)
{
	long failure = 0;
	double bound = pow(2.0, static_cast<double>(-precision));
	PhantomPlaintext plain_out;
	vector<double> output;
	ckksevaluator.decryptor.decrypt(cipher, plain_out);
	ckksevaluator.encoder.decode(plain_out, output);

	for (int i = 0; i < n; ++i)
		if (abs(ReLU(x[i]) - output[i]) > bound)
			failure++;

	cout << "-------------------------------------------------" << endl;
	cout << "failure : " << failure << endl;
	cout << "-------------------------------------------------" << endl;
	return failure;
}

void decrypt_and_print( PhantomCiphertext &cipher, CKKSEvaluator &ckksevaluator, long sparse_slots, size_t front, size_t back)
{
	PhantomPlaintext plain;
	ckksevaluator.decryptor.decrypt(cipher, plain);

	vector<complex<double>> rtn_vec;
	// encoder.decode(plain, rtn_vec, sparse_slots);
	ckksevaluator.encoder.decode(plain, rtn_vec);

	cout << "( ";
	for (size_t i = 0; i < front; i++)
		cout << rtn_vec[i] << ", ";
	cout << "... ";

	size_t slots;
	if (sparse_slots == 0)
		slots = rtn_vec.size();
	else
		slots = sparse_slots;
	for (size_t i = 0; i < back; i++)
	{
		cout << rtn_vec[slots - back + i];
		if (i != back - 1)
			cout << ", ";
	}
	cout << ")" << endl;
}
void decrypt_and_print_part( PhantomCiphertext &cipher, CKKSEvaluator &ckksevaluator, long sparse_slots, size_t start, size_t end)
{
	PhantomPlaintext plain;
	ckksevaluator.decryptor.decrypt(cipher, plain);

	vector<complex<double>> rtn_vec;
	// encoder.decode(plain, rtn_vec, sparse_slots);
	ckksevaluator.encoder.decode(plain, rtn_vec);

	cout << "( ";
	cout << "... ";
	for (size_t i = start; i <= end; i++)
		cout << rtn_vec[i] << ", ";
	cout << "... ";
	cout << ")" << endl;
	cout << "scale: " << cipher.scale() << endl;
}
void decrypt_and_print_txt( PhantomCiphertext &cipher, CKKSEvaluator &ckksevaluator, long sparse_slots, size_t front, size_t back, ofstream &output)
{
	PhantomPlaintext plain;
	ckksevaluator.decryptor.decrypt(cipher, plain);

	vector<complex<double>> rtn_vec;
	// encoder.decode(plain, rtn_vec, sparse_slots);
	ckksevaluator.encoder.decode(plain, rtn_vec);

	cout << "( ";
	output << "( ";
	for (size_t i = 0; i < front; i++)
	{
		cout << rtn_vec[i] << ", ";
		if ((i + 1) % 1024 == 0)
			cout << endl
				 << endl;
	}
	for (size_t i = 0; i < front; i++)
	{
		output << rtn_vec[i] << ", ";
		if ((i + 1) % 1024 == 0)
			output << endl
				   << endl;
	}
	cout << "... ";
	output << "... ";

	size_t slots;
	if (sparse_slots == 0)
		slots = rtn_vec.size();
	else
		slots = sparse_slots;
	for (size_t i = 0; i < back; i++)
	{
		cout << rtn_vec[slots - back + i];
		output << rtn_vec[slots - back + i];
		if (i != back - 1)
			cout << ", ";
	}
	cout << ")" << endl;
	output << ")" << endl;

	cout << endl;
	output << endl;
}

// infer_seal
//  import parameters
void import_parameters_cifar10(vector<double> &linear_weight, vector<double> &linear_bias, vector<vector<double>> &conv_weight, vector<vector<double>> &bn_bias, vector<vector<double>> &bn_running_mean, vector<vector<double>> &bn_running_var, vector<vector<double>> &bn_weight, size_t layer_num, size_t end_num)
{
	string dir;
	if (layer_num != 20 && layer_num != 32 && layer_num != 44 && layer_num != 56 && layer_num != 110)
		throw std::invalid_argument("layer number is not valid");
	if (layer_num == 20)
		dir = "resnet20_new";
	else if (layer_num == 32)
		dir = "resnet32_new";
	else if (layer_num == 44)
		dir = "resnet44_new";
	else if (layer_num == 56)
		dir = "resnet56_new";
	else if (layer_num == 110)
		dir = "resnet110_new";

	ifstream in;
	double val;
	size_t num_c = 0, num_b = 0, num_m = 0, num_v = 0, num_w = 0;

	conv_weight.clear();
	conv_weight.resize(layer_num - 1);
	bn_bias.clear();
	bn_bias.resize(layer_num - 1);
	bn_running_mean.clear();
	bn_running_mean.resize(layer_num - 1);
	bn_running_var.clear();
	bn_running_var.resize(layer_num - 1);
	bn_weight.clear();
	bn_weight.resize(layer_num - 1);

	int fh = 3, fw = 3;
	int ci = 0, co = 0;

	// convolution parameters
	ci = 3, co = 16;
	in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/conv1_weight.txt");
	
	if (!in.is_open())
		throw std::runtime_error("file is not open");
	for (long i = 0; i < fh * fw * ci * co; i++)
	{
		in >> val;
		conv_weight[num_c].emplace_back(val);
	}
	in.close();
	num_c++;

	// convolution parameters
	for (int j = 1; j <= 3; j++)
	{
		for (int k = 0; k <= end_num; k++)
		{
			// co setting
			if (j == 1)
				co = 16;
			else if (j == 2)
				co = 32;
			else if (j == 3)
				co = 64;

			// ci setting
			if (j == 1 || (j == 2 && k == 0))
				ci = 16;
			else if ((j == 2 && k != 0) || (j == 3 && k == 0))
				ci = 32;
			else
				ci = 64;
			in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/layer" + to_string(j) + "_" + to_string(k) + "_conv1_weight.txt");
			if (!in.is_open())
				throw std::runtime_error("file is not open");
			for (long i = 0; i < fh * fw * ci * co; i++)
			{
				in >> val;
				conv_weight[num_c].emplace_back(val);
			}
			in.close();
			num_c++;

			// ci setting
			if (j == 1)
				ci = 16;
			else if (j == 2)
				ci = 32;
			else if (j == 3)
				ci = 64;
			in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/layer" + to_string(j) + "_" + to_string(k) + "_conv2_weight.txt");
			if (!in.is_open())
				throw std::runtime_error("file is not open");
			for (long i = 0; i < fh * fw * ci * co; i++)
			{
				in >> val;
				conv_weight[num_c].emplace_back(val);
			}
			in.close();
			num_c++;
		}
	}

	// batch_normalization parameters
	ci = 16;
	in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/bn1_bias.txt");
	if (!in.is_open())
		throw std::runtime_error("file is not open");
	for (long i = 0; i < ci; i++)
	{
		in >> val;
		bn_bias[num_b].emplace_back(val);
	}
	in.close();
	num_b++;
	in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/bn1_running_mean.txt");
	if (!in.is_open())
		throw std::runtime_error("file is not open");
	for (long i = 0; i < ci; i++)
	{
		in >> val;
		bn_running_mean[num_m].emplace_back(val);
	}
	in.close();
	num_m++;
	in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/bn1_running_var.txt");
	if (!in.is_open())
		throw std::runtime_error("file is not open");
	for (long i = 0; i < ci; i++)
	{
		in >> val;
		bn_running_var[num_v].emplace_back(val);
	}
	in.close();
	num_v++;
	in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/bn1_weight.txt");
	if (!in.is_open())
		throw std::runtime_error("file is not open");
	for (long i = 0; i < ci; i++)
	{
		in >> val;
		bn_weight[num_w].emplace_back(val);
	}
	in.close();
	num_w++;

	// batch_normalization parameters
	for (int j = 1; j <= 3; j++)
	{
		int ci;
		if (j == 1)
			ci = 16;
		else if (j == 2)
			ci = 32;
		else if (j == 3)
			ci = 64;

		for (int k = 0; k <= end_num; k++)
		{
			in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/layer" + to_string(j) + "_" + to_string(k) + "_bn1_bias.txt");
			if (!in.is_open())
				throw std::runtime_error("file is not open");
			for (long i = 0; i < ci; i++)
			{
				in >> val;
				bn_bias[num_b].emplace_back(val);
			}
			in.close();
			num_b++;
			in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/layer" + to_string(j) + "_" + to_string(k) + "_bn1_running_mean.txt");
			if (!in.is_open())
				throw std::runtime_error("file is not open");
			for (long i = 0; i < ci; i++)
			{
				in >> val;
				bn_running_mean[num_m].emplace_back(val);
			}
			in.close();
			num_m++;
			in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/layer" + to_string(j) + "_" + to_string(k) + "_bn1_running_var.txt");
			if (!in.is_open())
				throw std::runtime_error("file is not open");
			for (long i = 0; i < ci; i++)
			{
				in >> val;
				bn_running_var[num_v].emplace_back(val);
			}
			in.close();
			num_v++;
			in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/layer" + to_string(j) + "_" + to_string(k) + "_bn1_weight.txt");
			if (!in.is_open())
				throw std::runtime_error("file is not open");
			for (long i = 0; i < ci; i++)
			{
				in >> val;
				bn_weight[num_w].emplace_back(val);
			}
			in.close();
			num_w++;
			in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/layer" + to_string(j) + "_" + to_string(k) + "_bn2_bias.txt");
			if (!in.is_open())
				throw std::runtime_error("file is not open");
			for (long i = 0; i < ci; i++)
			{
				in >> val;
				bn_bias[num_b].emplace_back(val);
			}
			in.close();
			num_b++;
			in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/layer" + to_string(j) + "_" + to_string(k) + "_bn2_running_mean.txt");
			if (!in.is_open())
				throw std::runtime_error("file is not open");
			for (long i = 0; i < ci; i++)
			{
				in >> val;
				bn_running_mean[num_m].emplace_back(val);
			}
			in.close();
			num_m++;
			in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/layer" + to_string(j) + "_" + to_string(k) + "_bn2_running_var.txt");
			if (!in.is_open())
				throw std::runtime_error("file is not open");
			for (long i = 0; i < ci; i++)
			{
				in >> val;
				bn_running_var[num_v].emplace_back(val);
			}
			in.close();
			num_v++;
			in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/layer" + to_string(j) + "_" + to_string(k) + "_bn2_weight.txt");
			if (!in.is_open())
				throw std::runtime_error("file is not open");
			for (long i = 0; i < ci; i++)
			{
				in >> val;
				bn_weight[num_w].emplace_back(val);
			}
			in.close();
			num_w++;
		}
	}

	// FC
	in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/linear_weight.txt");
	if (!in.is_open())
		throw std::runtime_error("file is not open");
	for (long i = 0; i < 10 * 64; i++)
	{
		in >> val;
		linear_weight.emplace_back(val);
	}
	in.close();
	in.open("/root/phantom-fhe/pretrained_parameters/" + dir + "/linear_bias.txt");
	if (!in.is_open())
		throw std::runtime_error("file is not open");
	for (long i = 0; i < 10; i++)
	{
		in >> val;
		linear_bias.emplace_back(val);
	}
	in.close();
}
