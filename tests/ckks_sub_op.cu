#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include "phantom.h"
#include <vector>
#include <cmath>
#include <random>
#include <memory>

using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;
using namespace std;

const double EPSILON = 0.001;

std::vector<complex<double>> generate_random_vector(size_t size) {
    std::vector<complex<double>> result(size);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-1.0, 1.0);
    for (size_t i = 0; i < size; ++i) {
        result[i] = complex<double>(dis(gen), dis(gen));
    }
    return result;
}

void run_sub_test(size_t poly_modulus_degree, const vector<int>& coeff_modulus, double scale){
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(phantom::arith::CoeffModulus::Create(poly_modulus_degree, coeff_modulus));

    PhantomContext context(parms);
    PhantomCKKSEncoder encoder(context);
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);
    PhantomRelinKey relin_keys = secret_key.gen_relinkey(context);
    PhantomGaloisKey galois_keys = secret_key.create_galois_keys(context);

    CKKSEvaluator ckks_evaluator(&context, &public_key, &secret_key, &encoder, &relin_keys, &galois_keys, scale);

    int slots = ckks_evaluator.encoder.slot_count();
    vector<complex<double>> input1_vector = generate_random_vector(slots);
    vector<complex<double>> input2_vector = generate_random_vector(slots);

    PhantomPlaintext plain1, plain2;
    ckks_evaluator.encoder.encode(input1_vector, scale, plain1);
    ckks_evaluator.encoder.encode(input2_vector, scale, plain2);

    //sub_inplace
    PhantomCiphertext cipher1_inplace, cipher2_inplace;
    ckks_evaluator.encryptor.encrypt(plain1, cipher1_inplace);
    ckks_evaluator.encryptor.encrypt(plain2, cipher2_inplace);

    ckks_evaluator.evaluator.sub_inplace(cipher1_inplace, cipher2_inplace);
    PhantomPlaintext subresult_inplace;
    ckks_evaluator.decryptor.decrypt(cipher1_inplace, subresult_inplace);
    vector<complex<double>> output_inplace;
    ckks_evaluator.encoder.decode(subresult_inplace, output_inplace);

    ASSERT_EQ(input1_vector.size(), output_inplace.size());
    for(size_t i = 0; i < input1_vector.size(); i++){
        EXPECT_NEAR(input1_vector[i].real() - input2_vector[i].real(), output_inplace[i].real(), EPSILON);
        EXPECT_NEAR(input1_vector[i].imag() - input2_vector[i].imag(), output_inplace[i].imag(), EPSILON);
    }

    //sub_plain
    PhantomCiphertext cipher1_plain, dest_plain;
    ckks_evaluator.encryptor.encrypt(plain1, cipher1_plain);
    ckks_evaluator.evaluator.sub_plain(cipher1_plain ,plain2, dest_plain); 

    PhantomPlaintext subresult_plain;
    ckks_evaluator.decryptor.decrypt(dest_plain, subresult_plain);

    vector<complex<double>> output_plain;
    ckks_evaluator.encoder.decode(subresult_plain, output_plain);

    ASSERT_EQ(input1_vector.size(), output_plain.size());
    for (size_t i = 0; i < input1_vector.size(); i++){
        EXPECT_NEAR(input1_vector[i].real() - input2_vector[i].real(), output_plain[i].real(), EPSILON);
        EXPECT_NEAR(input1_vector[i].imag() - input2_vector[i].imag(), output_plain[i].imag(), EPSILON);
    }

    //sub_plain_inplace
    PhantomCiphertext cipher1_plain_inplace;
    ckks_evaluator.encryptor.encrypt(plain1, cipher1_plain_inplace);    
    
    ckks_evaluator.evaluator.sub_plain_inplace(cipher1_plain_inplace, plain2);

    PhantomPlaintext subresult_plain_inplace;
    ckks_evaluator.decryptor.decrypt(cipher1_plain_inplace, subresult_plain_inplace);

    vector<complex<double>> output_plain_inplace;
    ckks_evaluator.encoder.decode(subresult_plain_inplace, output_plain_inplace);

    ASSERT_EQ(input1_vector.size(), output_plain_inplace.size());
    for (size_t i = 0; i < input1_vector.size(); i++){
        EXPECT_NEAR(input1_vector[i].real() - input2_vector[i].real(), output_plain_inplace[i].real(), EPSILON);
        EXPECT_NEAR(input1_vector[i].imag() - input2_vector[i].imag(), output_plain_inplace[i].imag(), EPSILON);
    }

    //sub
    PhantomCiphertext cipher1_sub, cipher2_sub, dest_sub;
    ckks_evaluator.encryptor.encrypt(plain1, cipher1_sub);
    ckks_evaluator.encryptor.encrypt(plain2, cipher2_sub);

    ckks_evaluator.evaluator.sub(cipher1_sub, cipher2_sub, dest_sub);

    PhantomPlaintext plain_sub;
    ckks_evaluator.decryptor.decrypt(dest_sub, plain_sub);
    vector<complex<double>> output_sub;
    ckks_evaluator.encoder.decode(plain_sub, output_sub);
    
    ASSERT_EQ(input1_vector.size(), output_sub.size());
    for (size_t i = 0; i < input1_vector.size(); i++){
        EXPECT_NEAR(input1_vector[i].real() - input2_vector[i].real(), output_sub[i].real(), EPSILON);
        EXPECT_NEAR(input1_vector[i].imag() - input2_vector[i].imag(), output_sub[i].imag(), EPSILON);
    }
}

namespace phantomtest{
    TEST(PhantomCKKSBasicOperationsTest, SubOperationTest1) {
        run_sub_test(65536, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60}, pow(2.0, 40));
    }
    TEST(PhantomCKKSBasicOperationsTest, SubOperationTest2) {
        run_sub_test(8192, {60, 30, 30, 30, 60}, pow(2.0, 30));
    }

    TEST(PhantomCKKSBasicOperationsTest, SubOperationTest3) {
        run_sub_test(16384, {60, 40, 40, 40, 40, 40, 40, 40, 60}, pow(2.0, 40));
    }

    TEST(PhantomCKKSBasicOperationsTest, SubOperationTest4) {
        run_sub_test(32768, {60, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 60}, pow(2.0, 50));
    }
    TEST(PhantomCKKSBasicOperationsTest, SubOperationTest5) {
        run_sub_test(65536, {60, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 60}, pow(2.0, 50));
    }
    TEST(PhantomCKKSBasicOperationsTest, SubOperationTest6) {
        run_sub_test(8192, {60, 40, 40, 60}, pow(2.0, 40));
    }
    TEST(PhantomCKKSBasicOperationsTest, SubOperationTest7) {
        run_sub_test(16384, {50, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 50}, pow(2.0, 30));
    }

    TEST(PhantomCKKSBasicOperationsTest, SubOperationTest8) {
        run_sub_test(32768, {60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60}, pow(2.0, 60));
    }
    TEST(PhantomCKKSBasicOperationsTest, SubOperationTest9) {
        run_sub_test(8192, {30, 30, 30, 30}, pow(2.0, 30));
    }
    TEST(PhantomCKKSBasicOperationsTest, SubOperationTest10) {
        run_sub_test(32768, {30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30}, pow(2.0, 30));
    }
}

