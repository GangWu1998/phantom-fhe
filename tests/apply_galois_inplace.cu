#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include "phantom.h"
#include "boot/Bootstrapper.cuh"
#include <vector>
#include <cmath>
#include <random>
#include <memory>
#include <Eigen/Dense>

using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;
using namespace std;

const double EPSILON = 0.1;

std::vector<complex<double>> generate_random_vector(size_t size) {
    std::vector<complex<double>> result(size);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-1.0, 1.0);
    for (size_t i = 0; i < size; ++i) {
        result[i] = complex<double>(dis(gen), dis(gen));
    }
    return result;
}
Eigen::VectorXcd vectorToEigen(const std::vector<std::complex<double>>& v) {
    Eigen::VectorXcd ev(v.size());
    for (size_t i = 0; i < v.size(); ++i) {
        ev[i] = v[i];
    }
    return ev;
}

void run_rotation_test(size_t poly_modulus_degree, const vector<int>& coeff_modulus, double scale){
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(phantom::arith::CoeffModulus::Create(poly_modulus_degree, coeff_modulus));

    PhantomContext context(parms);
    PhantomCKKSEncoder encoder(context);
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);
    PhantomRelinKey relin_keys = secret_key.gen_relinkey(context);
    PhantomGaloisKey galois_keys;

    CKKSEvaluator ckks_evaluator(&context, &public_key, &secret_key, &encoder, &relin_keys, &galois_keys, scale);

    int slots = ckks_evaluator.encoder.slot_count();
    vector<complex<double>> input_vector = generate_random_vector(slots);

    PhantomPlaintext plain, result;
    ckks_evaluator.encoder.encode(input_vector, scale, plain);

    PhantomCiphertext cipher_galois_inplace;
    ckks_evaluator.encryptor.encrypt(plain, cipher_galois_inplace);
    vector<int> steps = {1, 2, 3, 4, 5};
    ckks_evaluator.decryptor.create_galois_keys_from_steps(steps, galois_keys);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    ckks_evaluator.evaluator.apply_galois_inplace(cipher_galois_inplace, 3, galois_keys);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    //duration<double> sec = system_clock::now() - start;
    std::cout << "Sub Kernel execution time: " << elapsedTime * 1000 << " us" << std::endl;

    ckks_evaluator.decryptor.decrypt(cipher_galois_inplace, result);
    vector<complex<double>> output_vector;
    ckks_evaluator.encoder.decode(result, output_vector);

    ASSERT_EQ(input_vector.size(), output_vector.size());
    vector<complex<double>> eigen_input_vector(input_vector.size());
    for(int i = 0; i < input_vector.size(); ++i){
        eigen_input_vector[i] = input_vector[(i + 3) % input_vector.size()];
    }

    Eigen::VectorXcd input1_eigen = vectorToEigen(eigen_input_vector);
    Eigen::VectorXcd output_eigen = vectorToEigen(output_vector);
    Eigen::VectorXd absolute_error = (input1_eigen - output_eigen).cwiseAbs();
    Eigen::VectorXd relative_error = absolute_error.cwiseQuotient(input1_eigen.cwiseAbs());
 
    double mse = (input1_eigen - output_eigen).squaredNorm() / input1_eigen.size();
    std::cout << "Sub Mean Squared Error (MSE): " << mse << std::endl;

    double max_error = absolute_error.maxCoeff();
    std::cout << "Sub Max Error: " << max_error << std::endl;
}

namespace phantomtest{
    TEST(PhantomCKKSBasicOperationsTest, RotationOperationTest1) {
        run_rotation_test(8192, {60, 40, 40, 60}, pow(2.0, 40));
    }
    TEST(PhantomCKKSBasicOperationsTest, RotationOperationTest2) {
        run_rotation_test(8192, {50, 40, 40, 50}, pow(2.0, 40));
    }

    TEST(PhantomCKKSBasicOperationsTest, RotationOperationTest3) {
        run_rotation_test(16384, {60, 50, 50, 50, 50, 50, 50, 60}, pow(2.0, 50));
    }

    TEST(PhantomCKKSBasicOperationsTest, RotationOperationTest4) {
        run_rotation_test(16384, {60, 45, 45, 45, 45, 45, 45, 45, 60}, pow(2.0, 45));
    }
    TEST(PhantomCKKSBasicOperationsTest, RotationOperationTest5) {
        run_rotation_test(16384, {60, 40, 40, 40, 40, 40, 40, 40, 60}, pow(2.0, 40));
    }
    TEST(PhantomCKKSBasicOperationsTest, RotationOperationTest6) {
        run_rotation_test(32768, {60, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 60}, pow(2.0, 50));
    }
    TEST(PhantomCKKSBasicOperationsTest, RotationOperationTest7) {
        run_rotation_test(32768, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60}, pow(2.0, 40));
    }

    TEST(PhantomCKKSBasicOperationsTest, RotationOperationTest8) {
        run_rotation_test(32768, {60, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 60}, pow(2.0, 60));
    }
    TEST(PhantomCKKSBasicOperationsTest, RotationOperationTest9) {
        run_rotation_test(65536, {60, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 60}, pow(2.0, 60));
    }
    TEST(PhantomCKKSBasicOperationsTest, RotationOperationTest10) {
        run_rotation_test(65536, {60, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50,50, 60}, pow(2.0, 50));
    }
}

