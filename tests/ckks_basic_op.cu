#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include "phantom.h"
#include <vector>
#include <cmath>
#include <random>
#include <memory>

using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;

class PhantomCKKSBasicOperationsTest : public ::testing::Test {
protected:
    void SetUp() override {
        parms = std::make_unique<EncryptionParameters>(scheme_type::ckks);
        poly_modulus_degree = 8192;
        parms->set_poly_modulus_degree(poly_modulus_degree);
        parms->set_coeff_modulus(CoeffModulus::Create(poly_modulus_degree, {60, 40, 40, 60}));
        scale = pow(2.0, 40);
        
        context = std::make_unique<PhantomContext>(*parms);
        encoder = std::make_unique<PhantomCKKSEncoder>(*context);
        secret_key = std::make_unique<PhantomSecretKey>(*context);
        public_key = std::make_unique<PhantomPublicKey>(secret_key->gen_publickey(*context));
        relin_keys = std::make_unique<PhantomRelinKey>(secret_key->gen_relinkey(*context));
        galois_keys = std::make_unique<PhantomGaloisKey>(secret_key->create_galois_keys(*context));
    }

    std::unique_ptr<EncryptionParameters> parms;
    size_t poly_modulus_degree;
    std::unique_ptr<PhantomContext> context;
    std::unique_ptr<PhantomCKKSEncoder> encoder;
    std::unique_ptr<PhantomSecretKey> secret_key;
    std::unique_ptr<PhantomPublicKey> public_key;
    std::unique_ptr<PhantomRelinKey> relin_keys;
    std::unique_ptr<PhantomGaloisKey> galois_keys;
    double scale;

    const double EPSILON = 0.001;

    std::vector<hipDoubleComplex> generate_random_vector(size_t size) {
        std::vector<hipDoubleComplex> result(size);
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<> dis(-1.0, 1.0);
        for (size_t i = 0; i < size; ++i) {
            result[i] = make_hipDoubleComplex(dis(gen), dis(gen));
        }
        return result;
    }
};

TEST_F(PhantomCKKSBasicOperationsTest, EncodeDecodeTest) {
    std::vector<hipDoubleComplex> input = generate_random_vector(encoder->slot_count());
    
    PhantomPlaintext plain;
    encoder->encode(*context, input, scale, plain);
    
    std::vector<hipDoubleComplex> output;
    encoder->decode(*context, plain, output);
    
    ASSERT_EQ(input.size(), output.size());
    for (size_t i = 0; i < input.size(); i++) {
        EXPECT_NEAR(input[i].x, output[i].x, EPSILON);
        EXPECT_NEAR(input[i].y, output[i].y, EPSILON);
    }
}

TEST_F(PhantomCKKSBasicOperationsTest, SymmetricEncryptionTest) {
    std::vector<hipDoubleComplex> input = generate_random_vector(encoder->slot_count());
    
    PhantomPlaintext plain, decrypted_plain;
    encoder->encode(*context, input, scale, plain);
    
    PhantomCiphertext cipher;
    secret_key->encrypt_symmetric(*context, plain, cipher);
    
    secret_key->decrypt(*context, cipher, decrypted_plain);
    
    std::vector<hipDoubleComplex> output;
    encoder->decode(*context, decrypted_plain, output);
    
    ASSERT_EQ(input.size(), output.size());
    for (size_t i = 0; i < input.size(); i++) {
        EXPECT_NEAR(input[i].x, output[i].x, EPSILON);
        EXPECT_NEAR(input[i].y, output[i].y, EPSILON);
    }
}

TEST_F(PhantomCKKSBasicOperationsTest, HomomorphicAdditionTest) {
    std::vector<hipDoubleComplex> input1 = generate_random_vector(encoder->slot_count());
    std::vector<hipDoubleComplex> input2 = generate_random_vector(encoder->slot_count());
    
    PhantomPlaintext plain1, plain2, result_plain;
    encoder->encode(*context, input1, scale, plain1);
    encoder->encode(*context, input2, scale, plain2);
    
    PhantomCiphertext cipher1, cipher2;
    public_key->encrypt_asymmetric(*context, plain1, cipher1);
    public_key->encrypt_asymmetric(*context, plain2, cipher2);
    
    add_inplace(*context, cipher1, cipher2);
    
    secret_key->decrypt(*context, cipher1, result_plain);
    
    std::vector<hipDoubleComplex> output;
    encoder->decode(*context, result_plain, output);
    
    ASSERT_EQ(input1.size(), output.size());
    for (size_t i = 0; i < input1.size(); i++) {
        EXPECT_NEAR(input1[i].x + input2[i].x, output[i].x, EPSILON);
        EXPECT_NEAR(input1[i].y + input2[i].y, output[i].y, EPSILON);
    }
}

TEST_F(PhantomCKKSBasicOperationsTest, HomomorphicMultiplicationTest) {
    std::vector<hipDoubleComplex> input1 = generate_random_vector(encoder->slot_count());
    std::vector<hipDoubleComplex> input2 = generate_random_vector(encoder->slot_count());
    
    PhantomPlaintext plain1, plain2, result_plain;
    encoder->encode(*context, input1, scale, plain1);
    encoder->encode(*context, input2, scale, plain2);
    
    PhantomCiphertext cipher1, cipher2;
    public_key->encrypt_asymmetric(*context, plain1, cipher1);
    public_key->encrypt_asymmetric(*context, plain2, cipher2);
    
    PhantomCiphertext cipher_result = multiply(*context, cipher1, cipher2);
    relinearize_inplace(*context, cipher_result, *relin_keys);
    rescale_to_next_inplace(*context, cipher_result);
    
    secret_key->decrypt(*context, cipher_result, result_plain);
    
    std::vector<hipDoubleComplex> output;
    encoder->decode(*context, result_plain, output);
    
    ASSERT_EQ(input1.size(), output.size());
    for (size_t i = 0; i < input1.size(); i++) {
        hipDoubleComplex expected = hipCmul(input1[i], input2[i]);
        EXPECT_NEAR(expected.x, output[i].x, EPSILON * 10);
        EXPECT_NEAR(expected.y, output[i].y, EPSILON * 10);
    }
}

TEST_F(PhantomCKKSBasicOperationsTest, RotationTest) {
    std::vector<hipDoubleComplex> input(encoder->slot_count());
    for (size_t i = 0; i < input.size(); i++) {
        input[i] = make_hipDoubleComplex((double)i, 0.0);
    }
    
    PhantomPlaintext plain, result_plain;
    encoder->encode(*context, input, scale, plain);
    
    PhantomCiphertext cipher;
    public_key->encrypt_asymmetric(*context, plain, cipher);
    
    int rotation_steps = 3;
    rotate_inplace(*context, cipher, rotation_steps, *galois_keys);
    
    secret_key->decrypt(*context, cipher, result_plain);
    
    std::vector<hipDoubleComplex> output;
    encoder->decode(*context, result_plain, output);
    
    ASSERT_EQ(input.size(), output.size());
    for (size_t i = 0; i < input.size(); i++) {
        size_t expected_index = (i + rotation_steps) % input.size();
        EXPECT_NEAR(input[expected_index].x, output[i].x, EPSILON);
        EXPECT_NEAR(input[expected_index].y, output[i].y, EPSILON);
    }
}

