#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include "phantom.h"
#include "boot/Bootstrapper.cuh"
#include <vector>
#include <cmath>
#include <random>
#include <memory>
#include <Eigen/Dense>

using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;
using namespace std;

std::vector<complex<double>> generate_random_vector(size_t size) {
    std::vector<complex<double>> result(size);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-1.0, 1.0);
    for (size_t i = 0; i < size; ++i) {
        result[i] = complex<double>(dis(gen), dis(gen));
    }
    return result;
}
Eigen::VectorXcd vectorToEigen(const std::vector<std::complex<double>>& v) {
    Eigen::VectorXcd ev(v.size());
    for (size_t i = 0; i < v.size(); ++i) {
        ev[i] = v[i];
    }
    return ev;
}

void run_multi_test(size_t poly_modulus_degree, const vector<int>& coeff_modulus, double scale){
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(phantom::arith::CoeffModulus::Create(poly_modulus_degree, coeff_modulus));

    PhantomContext context(parms);
    PhantomCKKSEncoder encoder(context);
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);
    PhantomRelinKey relin_keys = secret_key.gen_relinkey(context);
    PhantomGaloisKey galois_keys = secret_key.create_galois_keys(context);

    CKKSEvaluator ckks_evaluator(&context, &public_key, &secret_key, &encoder, &relin_keys, &galois_keys, scale);

    int slots = ckks_evaluator.encoder.slot_count();
    vector<complex<double>> input1_vector = generate_random_vector(slots);
    vector<complex<double>> input2_vector = generate_random_vector(slots);

    PhantomPlaintext plain1, plain2;
    ckks_evaluator.encoder.encode(input1_vector, scale, plain1);
    ckks_evaluator.encoder.encode(input2_vector, scale, plain2);

    //multi_plain
    PhantomCiphertext cipher1_plain, dest_plain;
    ckks_evaluator.encryptor.encrypt(plain1, cipher1_plain);   

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    ckks_evaluator.evaluator.multiply_plain(cipher1_plain, plain2, dest_plain);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    //duration<double> sec = system_clock::now() - start;
    std::cout << "Sub Kernel execution time: " << elapsedTime * 1000 << " us" << std::endl;  

    PhantomPlaintext multiresult_plain;
    ckks_evaluator.decryptor.decrypt(dest_plain, multiresult_plain);

    vector<complex<double>> output_plain;
    ckks_evaluator.encoder.decode(multiresult_plain, output_plain);

    ASSERT_EQ(input1_vector.size(), output_plain.size());
    vector<complex<double>> input_vector(input1_vector.size()); 
    for(size_t i = 0; i < input1_vector.size(); i++){
        input_vector[i] = input1_vector[i] * input2_vector[i];
    }
    Eigen::VectorXcd input1_eigen = vectorToEigen(input_vector);
    Eigen::VectorXcd output_eigen = vectorToEigen(output_plain);
    Eigen::VectorXd absolute_error = (input1_eigen - output_eigen).cwiseAbs();
    Eigen::VectorXd relative_error = absolute_error.cwiseQuotient(input1_eigen.cwiseAbs());
 
    double mse = (input1_eigen - output_eigen).squaredNorm() / input1_eigen.size();
    std::cout << "Sub Mean Squared Error (MSE): " << mse << std::endl;

    double max_error = absolute_error.maxCoeff();
    std::cout << "Sub Max Error: " << max_error << std::endl;

}

namespace phantomtest{
    TEST(PhantomCKKSBasicOperationsTest, MultiOperationTest1) {
        run_multi_test(8192, {60, 40, 40, 60}, pow(2.0, 40));
    }
    TEST(PhantomCKKSBasicOperationsTest, MultiOperationTest2) {
        run_multi_test(8192, {50, 40, 40, 50}, pow(2.0, 40));
    }

    TEST(PhantomCKKSBasicOperationsTest, MultiOperationTest3) {
        run_multi_test(16384, {60, 50, 50, 50, 50, 50, 50, 60}, pow(2.0, 50));
    }

    TEST(PhantomCKKSBasicOperationsTest, MultiOperationTest4) {
        run_multi_test(16384, {60, 45, 45, 45, 45, 45, 45, 45, 60}, pow(2.0, 45));
    }
    TEST(PhantomCKKSBasicOperationsTest, MultiOperationTest5) {
        run_multi_test(16384, {60, 40, 40, 40, 40, 40, 40, 40, 60}, pow(2.0, 40));
    }
    TEST(PhantomCKKSBasicOperationsTest, MultiOperationTest6) {
        run_multi_test(32768, {60, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 60}, pow(2.0, 50));
    }
    TEST(PhantomCKKSBasicOperationsTest, MultiOperationTest7) {
        run_multi_test(32768, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60}, pow(2.0, 40));
    }

    TEST(PhantomCKKSBasicOperationsTest, MultiOperationTest8) {
        run_multi_test(32768, {60, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 60}, pow(2.0, 60));
    }
    TEST(PhantomCKKSBasicOperationsTest, MultiOperationTest9) {
        run_multi_test(65536, {60, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 55, 60}, pow(2.0, 60));
    }
    TEST(PhantomCKKSBasicOperationsTest, MultiOperationTest10) {
        run_multi_test(65536, {60, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50,50, 60}, pow(2.0, 50));
    }
}

